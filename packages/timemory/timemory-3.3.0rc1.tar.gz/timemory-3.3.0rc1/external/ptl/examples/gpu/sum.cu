#include "hip/hip_runtime.h"
//
// MIT License
// Copyright (c) 2018 Jonathan R. Madsen
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software. THE SOFTWARE IS PROVIDED
// "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT
// LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR
// PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN
// ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION
// WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
//
// ---------------------------------------------------------------
//
//   PTL CUDA implementation
//
//

//============================================================================//

#include "sum.hh"

#define PRINT_HERE(extra)                                                                \
    printf("> %s@'%s':%i %s\n", __FUNCTION__, __FILE__, __LINE__, extra)

//============================================================================//

//  gridDim:    This variable contains the dimensions of the grid.
//  blockIdx:   This variable contains the block index within the grid.
//  blockDim:   This variable and contains the dimensions of the block.
//  threadIdx:  This variable contains the thread index within the block.

//============================================================================//
//
//  efficient reduction
//  https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
//
//============================================================================//

template <unsigned int blockSize, typename _Tp>
__device__ void
warpReduce(volatile _Tp* _data, unsigned int tid)
{
    if(blockSize >= 64)
        _data[tid] += _data[tid + 32];
    if(blockSize >= 32)
        _data[tid] += _data[tid + 16];
    if(blockSize >= 16)
        _data[tid] += _data[tid + 8];
    if(blockSize >= 8)
        _data[tid] += _data[tid + 4];
    if(blockSize >= 4)
        _data[tid] += _data[tid + 2];
    if(blockSize >= 2)
        _data[tid] += _data[tid + 1];
}

//----------------------------------------------------------------------------//

template <unsigned int blockSize, typename _Tp>
__global__ void
reduce(_Tp* _idata, _Tp* _odata, unsigned int n)
{
    extern __shared__ _Tp _data[];
    unsigned int          tid      = threadIdx.x;
    unsigned int          i        = (2 * blockSize) * blockIdx.x + tid;
    unsigned int          gridSize = 2 * blockSize * gridDim.x;
    _data[tid]                     = 0;

    while(i < n)
    {
        _data[tid] += _idata[i] + _idata[i + blockSize];
        i += gridSize;
    }

    __syncthreads();

    if(blockSize >= 512)
    {
        if(tid < 256)
        {
            _data[tid] += _data[tid + 256];
        }
        __syncthreads();
    }

    if(blockSize >= 256)
    {
        if(tid < 128)
        {
            _data[tid] += _data[tid + 128];
        }
        __syncthreads();
    }

    if(blockSize >= 128)
    {
        if(tid < 64)
        {
            _data[tid] += _data[tid + 64];
        }
        __syncthreads();
    }

    if(tid < 32)
        warpReduce<blockSize, _Tp>(_data, tid);

    if(tid == 0)
        _odata[blockIdx.x] = _data[0];
}

//----------------------------------------------------------------------------//

template <typename _Tp>
void
compute_reduction(int threads, _Tp* _idata, _Tp* _odata, int dimGrid, int dimBlock,
                  int smemSize, hipStream_t stream)
{
    hipStreamSynchronize(stream);
    CUDA_CHECK_LAST_ERROR();

    switch(threads)
    {
        case 512:
            reduce<512, _Tp>
                <<<dimGrid, dimBlock, smemSize, stream>>>(_idata, _odata, threads);
            break;
        case 256:
            reduce<256, _Tp>
                <<<dimGrid, dimBlock, smemSize, stream>>>(_idata, _odata, threads);
            break;
        case 128:
            reduce<128, _Tp>
                <<<dimGrid, dimBlock, smemSize, stream>>>(_idata, _odata, threads);
            break;
        case 64:
            reduce<64, _Tp>
                <<<dimGrid, dimBlock, smemSize, stream>>>(_idata, _odata, threads);
            break;
        case 32:
            reduce<32, _Tp>
                <<<dimGrid, dimBlock, smemSize, stream>>>(_idata, _odata, threads);
            break;
        case 16:
            reduce<16, _Tp>
                <<<dimGrid, dimBlock, smemSize, stream>>>(_idata, _odata, threads);
            break;
        case 8:
            reduce<8, _Tp>
                <<<dimGrid, dimBlock, smemSize, stream>>>(_idata, _odata, threads);
            break;
        case 4:
            reduce<4, _Tp>
                <<<dimGrid, dimBlock, smemSize, stream>>>(_idata, _odata, threads);
            break;
        case 2:
            reduce<2, _Tp>
                <<<dimGrid, dimBlock, smemSize, stream>>>(_idata, _odata, threads);
            break;
        case 1:
            reduce<1, _Tp>
                <<<dimGrid, dimBlock, smemSize, stream>>>(_idata, _odata, threads);
            break;
    }
    CUDA_CHECK_LAST_ERROR();

    hipStreamSynchronize(stream);
    CUDA_CHECK_LAST_ERROR();
}

//============================================================================//

template <typename _Tp>
void
call_compute_reduction(int64_t& _i, uint64_t& _offset, int nthreads, _Tp* _idata,
                       _Tp* _odata, int dimGrid, int dimBlock, int smemSize,
                       hipStream_t stream)
{
    // assumes nthreads < cuda_max_threads_per_block()
    compute_reduction(nthreads, _idata + _offset, _odata + _offset, dimGrid, dimBlock,
                      smemSize, stream);
    _i -= nthreads;
    _offset += nthreads;
}

//============================================================================//

float
compute_sum_host(aligned_ptr<float>& data, hipStream_t stream, bool with_thrust,
                 float* buffer)
{
    float _sum;

    if(with_thrust)
    {
        hipStreamSynchronize(stream);
        CUDA_CHECK_LAST_ERROR();

        _sum = thrust::reduce(thrust::system::cuda::par.on(stream), data.ptr,
                              data.ptr + data.size, 0.0f, thrust::plus<float>());

        CUDA_CHECK_LAST_ERROR();

        hipStreamSynchronize(stream);
        CUDA_CHECK_LAST_ERROR();
    }
    else
    {
        // PRINT_HERE("");
        // PRINT_HERE(std::string(std::string("size    : ") +
        // std::to_string(data.size)).c_str());
        // PRINT_HERE(std::string(std::string("padding : ") +
        // std::to_string(data.padding)).c_str());
        // PRINT_HERE(std::string(std::string("storage : ") +
        // std::to_string(data.storage_size)).c_str());

        if(data.size < 1 || data.storage_size < 1)
            return 0.0f;

        int64_t  remain = data.size;
        uint64_t offset = 0;

        int smemSize = cuda_shared_memory_per_block();
        int dimGrid  = cuda_multi_processor_count();
        int dimBlock = cuda_max_threads_per_block();

        float* _idata = data.ptr;
        float* _odata = buffer;
        async_gpu_memset<float>(_odata, data.storage_size, stream);

        CUDA_CHECK_LAST_ERROR();

        while(remain > 0)
        {
            for(const auto& itr : { 512, 256, 128, 64, 32, 16, 8, 4, 2, 1 })
            {
                if(remain >= itr)
                {
                    call_compute_reduction(remain, offset, itr, _idata, _odata, dimGrid,
                                           dimBlock, smemSize, stream);
                    break;
                }
            }
        }

        hipMemcpyAsync(&_sum, _odata, 1 * sizeof(float), hipMemcpyDeviceToHost, stream);
        CUDA_CHECK_LAST_ERROR();
        hipDeviceSynchronize();
        CUDA_CHECK_LAST_ERROR();
    }

    return _sum;
}

//============================================================================//
